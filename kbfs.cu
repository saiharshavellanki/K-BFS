#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

using namespace std;

//For nodes in adjacency list
typedef struct node {
	int val;
	struct node* next;
}node;

//Stores visit array's old and new values
typedef struct node1
{
	int oldval,newval;
}node1;


//Compare function to sort based on decreasing order of oldvalue
int cmpfunc(const void* a,const void* b)
{
	node1 x,y;
	x = *(node1*)a;
	y = *(node1*)b;
	if(x.oldval<y.oldval)
		return 1;
	else if(x.oldval==y.oldval)
		return 0;
	else
		return -1;
}


//Function to update depth of nodes in next level of k-bfs
__global__
void Updatenextlevel(int *d_g_edges,int *d_g_edgepos,int *d_depth,node1 *d_visit,int *d_n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	for(i=index;i<d_n[0];i+=stride)
	{
		if(d_visit[i].oldval==d_depth[0])
		{
			for(j = d_g_edgepos[i];j<d_g_edgepos[i+1];j++)
			{
				if(d_visit[d_g_edges[j]].oldval==-1)
					d_visit[d_g_edges[j]].newval = d_depth[0]+1;
			}
		}
	}
}

//Function to update visit values after completion of iteration of k-bfs
__global__
void UpdateVisit(node1 *visit,int *d_n,int *d_depth,int *d_sz)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for(int j=index;j<d_n[0];j+=stride)
	{
		if(visit[j].newval==d_depth[0]+1)
			d_sz[0]+=1;
		visit[j].oldval = visit[j].newval;
    visit[j].newval = 0;
	}
}

//Array of vectors to store input graph in host memory
node *head[1157828];

//Inserting edge a->b in graph
void insert(int a,int b)
{
	node* temp;
	temp=(node*)malloc(sizeof(node));
	temp->val = b;
	temp->next = head[a];
	head[a] = temp;
}

int main()
{
	int n,m,i,a,b,start,end,j,k,q,t,K,num_blocks=1,num_threads=64;
	scanf("%d %d",&n,&m);

  //Initialising all the edge lists as NULL
	for(i=0;i<n+1;i++)
		head[i]=NULL;

  //Scanning Input graph
	for(i=0;i<m;i++)
	{
		scanf("%d %d",&a,&b);
		insert(a,b);
		insert(b,a);
	}

  //Value of K in K-bfs algorithm
	K = 32;

  /*G_edges is used to store edge end points in CSR format (stored in host memory)
  G_edgepos is used to store number of edges from vertex (stored in host memory)
  G_ecc to store our approximated eccentricity (stored in host memory)
  visit to store whether a vertex is visited (stored in host memory)*/
	int *G_edges,*G_edgepos,*G_ecc;
	node1 *visit;
	G_ecc = (int*)malloc((n+1)*sizeof(int));
	visit = (node1*)malloc((n+1)*sizeof(node1));
	G_edges = (int*)malloc((2*m)*sizeof(int));
	G_edgepos=(int*) malloc((n+2)*sizeof(int));
	G_edgepos[0] = 0 ;
	G_edgepos[1] = 0 ;

  //Converting graph to CSR format
	j = 0;
	for(i=1;i<=n;i++)
	{
		G_ecc[i] = 0;
		visit[i].oldval = -1;
		visit[i].newval = -1;
		node *temp = head[i];
		start = j;
		while(temp!=NULL)
		{
			G_edges[j] = temp->val;
			j++;
			temp=temp->next;
		}
		end = j-1;
		G_edgepos[i+1] = G_edgepos[i]+(end-start+1);
	}


  /*d_g_edges is used to store edge end points in CSR format (stored in device memory)
  d_g_edgepos is used to store number of edges from vertex (stored in device memory)
  d_g_ecc to store our approximated eccentricity (stored in device memory)
  d_visit to store whether a vertex is visited (stored in device memory)*/
	int *d_g_edges,*d_g_edgepos,*d_sz,*d_n;
	node1 *d_visit;

  /*Memory Allocation for variables in device memory and copying corresponding variables from
    host memory to device memory  */
	hipMalloc( (void**) &d_g_edges, sizeof(int)*(2*m)) ;
	hipMemcpy( d_g_edges,G_edges, sizeof(int)*(2*m), hipMemcpyHostToDevice) ;


	hipMalloc( (void**) &d_g_edgepos, sizeof(int)*(n+2)) ;
	hipMemcpy( d_g_edgepos,G_edgepos, sizeof(int)*(n+2), hipMemcpyHostToDevice) ;

	hipMalloc( (void**) &d_visit, sizeof(node1)*(n+1)) ;

	int *d_depth;
	hipMalloc( (void**) &d_depth, sizeof(int)) ;

	hipMalloc( (void**) &d_sz, sizeof(int)) ;

	hipMalloc( (void**) &d_n, sizeof(int)) ;
	hipMemcpy( d_n, &n, sizeof(int), hipMemcpyHostToDevice) ;


	for(i=1;i<=n;i++)
	{
    /*If vertex is not visited yet, then approximate the eccentricity
     of all vertices to which this vertex belongs.This vertex is taken as source vertex */
		if(G_ecc[i]==0)
		{
      //If a vertex is isolated vertex
			if(head[i]==NULL)
				continue;

      //Initialising value of K and visit array (host memory) before bfs
			k = K;
			for(j=1;j<=n;j++)
			{
				visit[j].oldval=-1;
				visit[j].newval=-1;
			}
      //Mark visit of first vertex found with zero approximated eccentricity as zero
			visit[i].oldval=0;

      //Copying host visit array to device visit array
			hipMemcpy(d_visit,visit, sizeof(node1)*(n+1), hipMemcpyHostToDevice) ;
			int sz=1,depth=0,comp_size=0;
			hipMemcpy(d_depth, &depth, sizeof(int), hipMemcpyHostToDevice) ;


      //Loop runs bfs on source vertex
      //The condition in while loop means we will stop when there are no nodes in current level of bfs
			while(sz>0)
			{
				comp_size+=sz;

        //Update next level in k-bfs
				Updatenextlevel<<<num_blocks,num_threads>>>(d_g_edges,d_g_edgepos,d_depth,d_visit,d_n);
				sz=0;

        //Copy size variable (which is 0) from host to device
				hipMemcpy(d_sz, &sz, sizeof(int), hipMemcpyHostToDevice) ;

        //Update visit array in device memory
				UpdateVisit<<<num_blocks,num_threads>>>(d_visit,d_n,d_depth,d_sz);

        //Increase the depth (host memory) in bfs
				depth++;

        //Copy depth variable in host memory to device memory
				hipMemcpy(d_depth, &depth, sizeof(int), hipMemcpyHostToDevice) ;

        //Copy the number of nodes in current level from device memory to host memory
				hipMemcpy(&sz, d_sz, sizeof(int), hipMemcpyDeviceToHost) ;
			}

      //Copying the visit array which has distances from the source back to host memory
			int *comp;
			hipMemcpy(visit,d_visit, (n+1)*sizeof(node1), hipMemcpyDeviceToHost) ;
			int l=0;

      //Getting number of nodes in current component
			for(j=1;j<=n;j++)
			{
				if(visit[j].oldval!=-1)
				{
					l++;
				}
			}

      //Adding values of nodes in current component to comp array (host memory)
			comp_size = l;
			comp = (int*)malloc(l*sizeof(int));
			l = 0;
			for(j=1;j<=n;j++)
			{
				if(visit[j].oldval!=-1)
				{
					comp[l]=j;
					l++;
				}
			}

      //If component size is less than k then k is changed to component size
			if(comp_size<k);
			k=comp_size;

      //Selecting k random nodes from component array
			for(j=0;j<k;j++)
			{
				q=rand()%comp_size;
				t = comp[j];
				comp[j]=comp[q];
				comp[q] = t;
			}

      //Initialise visit for all vertices as -1
			for(j=1;j<=n;j++)
			{
				visit[j].oldval = -1;
				visit[j].newval = -1;
			}

      //Mark visit for all vertices in currentcomponent as 0
			for(j=0;j<k;j++)
			{
				visit[comp[j]].oldval = 0;
				visit[comp[j]].newval = 0;
			}

      //Copy visit array from device memory to host memory
			hipMemcpy(d_visit,visit, sizeof(node1)*(n+1), hipMemcpyHostToDevice) ;

      //Initialise number of nodes in 1st level as k and their depth as 0
			sz=k,depth=0;

      //Copy depth variable from host memory to device memory
			hipMemcpy(d_depth,&depth, sizeof(int), hipMemcpyHostToDevice) ;

      //Running bfs with above selected k nodes in first level
			while(sz>0)
			{
        //Update next level in k-bfs
				Updatenextlevel<<<num_blocks,num_threads>>>(d_g_edges,d_g_edgepos,d_depth,d_visit,d_n);
				sz=0;

        //Copy size variable (which is 0) from host to device
				hipMemcpy(d_sz,&sz, sizeof(int), hipMemcpyHostToDevice) ;

        //Update visit array in device memory
				UpdateVisit<<<num_blocks,num_threads>>>(d_visit,d_n,d_depth,d_sz);

        //Increase the depth (host memory) in bfs
				depth++;

        //Copy depth variable in host memory to device memory
				hipMemcpy(d_depth,&depth, sizeof(int), hipMemcpyHostToDevice) ;

        //Copy the number of nodes in current level from device memory to host memory
				hipMemcpy(&sz,d_sz, sizeof(int), hipMemcpyDeviceToHost) ;
			}

      //Copying the visit array which has distances from the source back to host memory
			hipMemcpy(visit,d_visit, (n+1)*sizeof(node1), hipMemcpyDeviceToHost) ;

      //Update the eccentricities of nodes in this component based on visit array
			for(j=0;j<comp_size;j++)
			{
				G_ecc[comp[j]] = visit[comp[j]].oldval;
			}

      //newd array of type struct node1 which stores depth along with node value
			node1* newd;
			newd = (node1*)malloc(sizeof(node1));
			for(j=0;j<comp_size;j++)
			{
				newd[j].oldval = visit[comp[j]].oldval;
				newd[j].newval = comp[j];
			}

      //Sort newd array based on decreasing order of depth
			qsort(newd,comp_size,sizeof(node1),cmpfunc);

      //Initialise visit for all vertices as -1
			for(j=1;j<=n;j++)
			{
				visit[j].oldval=-1;
				visit[j].newval=-1;
			}

      //Mark visit for all vertices in currentcomponent as 0
			for(j=0;j<k;j++)
			{
				visit[newd[j].newval].oldval=0;
				visit[newd[j].newval].newval=0;
			}
      //Copy visit array from device memory to host memory
      hipMemcpy(d_visit,visit, sizeof(node1)*(n+1), hipMemcpyHostToDevice) ;

      //Initialise number of nodes in 1st level as k and their depth as 0
      sz=k,depth=0;

      //Copy depth variable from host memory to device memory
      hipMemcpy(d_depth,&depth, sizeof(int), hipMemcpyHostToDevice) ;

      //Running bfs with above selected k nodes in first level
      while(sz>0)
      {
        //Update next level in k-bfs
        Updatenextlevel<<<num_blocks,num_threads>>>(d_g_edges,d_g_edgepos,d_depth,d_visit,d_n);
        sz=0;

        //Copy size variable (which is 0) from host to device
        hipMemcpy(d_sz,&sz, sizeof(int), hipMemcpyHostToDevice) ;

        //Update visit array in device memory
        UpdateVisit<<<num_blocks,num_threads>>>(d_visit,d_n,d_depth,d_sz);

        //Increase the depth (host memory) in bfs
        depth++;

        //Copy depth variable in host memory to device memory
        hipMemcpy(d_depth,&depth, sizeof(int), hipMemcpyHostToDevice) ;

        //Copy the number of nodes in current level from device memory to host memory
        hipMemcpy(&sz,d_sz, sizeof(int), hipMemcpyDeviceToHost) ;
      }

      //Copying the visit array which has distances from the source back to host memory
      hipMemcpy(visit,d_visit, (n+1)*sizeof(node1), hipMemcpyDeviceToHost) ;

      /*Compare the value in visit array(depth)
      with previous approximated eccentricity value and update it if it is more*/
			for(j=0;j<comp_size;j++)
			{
				if(visit[comp[j]].oldval>G_ecc[comp[j]])
					G_ecc[comp[j]] = visit[comp[j]].oldval;
			}

		}

	}
	return 0;
}
